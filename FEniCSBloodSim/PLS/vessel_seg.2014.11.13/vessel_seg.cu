#include "hip/hip_runtime.h"
/* (C) 2014 F. Arguello, D.L. Vilarino, D.B. Heras and A. Nieto 
 *     CITIUS - Dept. of Electronics and Computing
 *     University of Santiago de Compostela 
 *
 * This program is licensed under a Creative Commons 
 * Attribution-NonCommercial-NoDerivatives 4.0 International License. 
 * ( http://creativecommons.org/licenses/by-nc-nd/4.0/ )
*/

// Code related to the paper:
//   F. Arguello, D.L. Vilarino, D.B. Heras and A. Nieto 
//   GPU-Based Segmentation of Retinal Blood Vessels
//   Journal of Real-Time Image Processing, 2014
//   DOI: 10.1007/s11554-014-0469-z
// input: image files in format PPM (color) and PGM (gray)
//   image collections, download from:
//   DRIVE: http://www.isi.uu.nl/Research/Databases/DRIVE/
//   STARE: http://www.ces.clemson.edu/~ahoover/stare/
//   MESSIDOR: http://messidor.crihan.fr/download.php
// output: vessel segmentations
// tested: Linux (CUDA version 4.2)
// compile with: nvcc -arch=sm_20 vessel_seg.cu -O3 -lm

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>
#include <fcntl.h>
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((a)>(b)?(a):(b))
#define BASE unsigned char
#define CONN 8
#define SOBEL_UMBRAL 0.5

// Selection of Image collection
#define DRIVE 0          // Download images from drive page
#define STARE 1          // Download images from stare page
#define MESSIDOR 2       // Download images from messidor page
#define COLLECTION DRIVE // Select here: DRIVE, STARE, or MESSIDOR

// Main parameters to be selected
#define THRESHOLD 4.32      // threshold for binarization of images
#define NOISE_SIZE 4        // pixel groups < 4 will be removed
#define SMALL_STRUC_SIZE 50 // small elonged structures will be removed

// Specific parameters DRIVE
#if COLLECTION==DRIVE
#define H 565          // horizontal size
#define V 584          // vertical size
#define STAGES 1       // PLS number of stages
#define TOPHAT 1       // 0=none, 1=circle r=17 2=square 15x15 3=square 128x128
#define FOV_UMBRAL 20  // gray level of pixels outside FOV <= FOV_UMBRAL
#define NIMG 20        // number of images in collection
#define PREFIX "images/drive/%s.ppm" // subdirectory localization of images
char name[NIMG][60], ground[NIMG][60], mask[NIMG][60]; 
void collect_images() 
{  int i;
   for(i=0;i<NIMG;i++) sprintf(name[i],"%02d_test",i+1);      // image names 
   for(i=0;i<NIMG;i++) sprintf(ground[i],"%02d_manual1",i+1); // ground truth
   for(i=0;i<NIMG;i++) sprintf(mask[i],"%02d_test_mask",i+1); // mask for FOV
}

// Specific parameters STARE
#elif COLLECTION==STARE
#define H 700         // horizontal size
#define V 605         // vertical size
#define STAGES 1      // PLS number of stages
#define TOPHAT 1      // 0=none, 1=circle r=17 2=square 15x15 3=square 128x128
#define FOV_UMBRAL 25 // gray level of pixels outside FOV <= FOV_UMBRAL
#define NIMG 20       // number of images in collection
#define PREFIX "images/stare/%s.ppm" // subdirectory localization of images
char name[NIMG][60], ground[NIMG][60], mask[NIMG][60]; 
void collect_images() 
{  int i, num[]={1,2,3,4,5,44,77,81,82,139,162,163,235,236,239,240,255,291,319,324};
   for(i=0;i<NIMG;i++) sprintf(name[i],"im%04d",num[i]);      // image names
   for(i=0;i<NIMG;i++) sprintf(ground[i],"im%04d.ah",num[i]); // ground truth
   for(i=0;i<NIMG;i++) sprintf(mask[i],"im%04d_mask",num[i]); // mask for FOV
}

// Specific parameters MESSIDOR
#elif COLLECTION==MESSIDOR
#define SMALL_STRUC_SIZE 200
#define H 2240        // horizontal size
#define V 1488        // vertical size
#define STAGES 1      // PLS number of stages
#define TOPHAT 3      // 0=none, 1=circle r=17 2=square 15x15 3=square 128x128
#define FOV_UMBRAL 20 // gray level of pixels outside FOV <= FOV_UMBRAL
#define NIMG 20       // number of images in collection
#define PREFIX "images/messidor/%s.ppm" // subdirectory localization of images
char name[NIMG][60], ground[NIMG][60], mask[NIMG][60]; 
void collect_images()
{  int i, num[]={38557,43808,43832,43882,43906,44261,44284,44338,44349,44400,44431,44598,44636,44692,44714,44762,44782,44843,44901,44923};
   for(i=0;i<1;i++) sprintf(name[i],"20051019_%05d_0100_PP",num[i]);  // image names
   for(i=1;i<20;i++) sprintf(name[i],"20051020_%05d_0100_PP",num[i]); // image names
}
#endif

// STADISTICS
double t_comp[NIMG], t_tot[NIMG], sens[NIMG], esp[NIMG], MAA[NIMG]; 

// -----------------------------------------------------------------------------
// IMAGES load/write, accuracies, averages
// -----------------------------------------------------------------------------

void read_image_ppm(const char *nombre, BASE img[V*H])
{  FILE* fp; int i, Nh, Nv, flag; unsigned char texto[100], *data, c;
   data=(unsigned char *) malloc(3*H*V*sizeof(char));
   if(data==NULL) { fprintf(stderr,"No enough RAM\n"); exit(-1); }
   fp = fopen (nombre, "rb");
   if(fp==NULL) { fprintf(stderr,"Can not open file %s\n",nombre); exit(1); }
   else printf("Open file %s,",nombre);
   flag=fscanf(fp,"%s",texto);
   if(strcasecmp((const char *)texto,"P6")) { fprintf(stderr,"No PPM file\n"); exit(-1); } 
   c=fgetc(fp); c=fgetc(fp);
   while((c=='#')) { while (c != '\n') c=fgetc(fp); c=fgetc(fp);}  ungetc(c,fp);
   flag=fscanf(fp,"%s",texto); Nh=atoi((const char*)texto); 
   flag=fscanf(fp,"%s",texto); Nv=atoi((const char*)texto);
   printf(" size %dx%d\n",Nh,Nv);
   if((Nh!=H)||(Nv!=V)) { fprintf(stderr,"Incorrect size image\n"); exit(-1); }
   flag=fscanf(fp,"%s",texto); c = fgetc(fp); ;
   flag=fread(data,3*H*V,1,fp); printf(" Read %u bytes\n",(unsigned int) flag*3*H*V);
     if(flag!=1) { fprintf(stderr,"Can not read all data\n"); exit(-1); }
   for(i=0;i<H*V;i++) img[i]=data[3*i+1];
   fclose(fp); free(data);
}

void read_image_ppm_cache(const char *nombre)
{  FILE* fp; int Nh, Nv, flag; unsigned char texto[100], *data, c;
   data=(unsigned char *) malloc(3*H*V*sizeof(char));
   if(data==NULL) { perror("No enough RAM\n"); exit(-1); }
   fp = fopen (nombre, "rb");
   if(fp==NULL) { fprintf(stderr,"Can not open file %s\n",nombre); exit(1); }
   flag=fscanf(fp,"%s",texto);   
   if(strcasecmp((const char *)texto,"P6")) { fprintf(stderr,"No PPM file\n"); exit(-1); } 
   c=fgetc(fp); c=fgetc(fp);
   while((c=='#')) { while (c != '\n') c = fgetc(fp); c = fgetc(fp); } ungetc(c,fp);
   flag=fscanf(fp,"%s",texto); Nh=atoi((const char*)texto); 
   flag=fscanf(fp,"%s",texto); Nv=atoi((const char*)texto);
   if((Nh!=H)||(Nv!=V)) { printf("Incorrect image size\n"); exit(-1); }
   flag=fscanf(fp,"%s",texto); c = fgetc(fp); ;
   flag=fread(data,3*H*V,1,fp); 
     if(flag!=1) { perror("Can not read all data"); exit(-1); }
   fclose(fp); free(data);
}

void read_image_pgm(const char *nombre, BASE img[V*H])
{  FILE* fp; int flag, Nh, Nv; unsigned char c, texto[100];
   fp = fopen (nombre, "rb");
   if(fp==NULL) { fprintf(stderr,"Can not open file %s\n",nombre); exit(1); }
   else printf("Open file %s,",nombre);
   flag=fscanf(fp,"%s",texto);   
   if(strcasecmp((const char *)texto,"P5")) { fprintf(stderr,"No PGM file\n"); exit(-1); } 
   c=fgetc(fp); c=fgetc(fp); 
   while((c=='#')) { while (c != '\n') c = fgetc(fp); c = fgetc(fp);} ungetc(c,fp);
   flag=fscanf(fp,"%s",texto); Nh=atoi((const char*)texto); 
   flag=fscanf(fp,"%s",texto); Nv=atoi((const char*)texto);
   printf(" size %dx%d\n",Nh,Nv);
   if((Nh!=H)||(Nv!=V)) { fprintf(stderr,"Incorrect size\n"); exit(-1); }
   flag=fscanf(fp,"%s",texto); c = fgetc(fp); ;
   flag=fread(img,H*V,1,fp); printf(" Read %u bytes\n",(unsigned int) flag*H*V);
   fclose(fp);
}

void save_image_pgm(const char fichero[40], BASE img[H*V])
{  FILE *fp; int i, flag; unsigned char *tmp;
   tmp=(unsigned char *)malloc(H*V*sizeof(char));
   if(tmp==NULL) { printf("No enough RAM\n"); exit(-1); }
   fp=fopen(fichero,"wb+");
   if(fp==NULL) { fprintf(stderr,"Can not save %s\n",fichero); exit(1); }
   else printf("Saved file %s, size %dx%d\n",fichero,H,V); 
   fprintf(fp,"P5\n#CREADOR University of Santiago de Compostela\n%d %d\n%d\n",H,V,255);
   for(i=0;i<H*V;i++) tmp[i]=255*img[i];
   flag=fwrite(tmp,H*V,1,fp); printf(" Saved %u bytes\n",(unsigned int) flag*H*V);
   fclose(fp); free(tmp);
}

void compute_accuracies(BASE *output, BASE *truth, int nimg)
{  char fichero[60]; int i, a00=0, a01=0, a10=0, a11=0, total; 
   BASE *mask_img;
   mask_img=(BASE *)malloc(H*V*sizeof(BASE));
   if(COLLECTION==DRIVE) sprintf(fichero,"images/drive/%s.pgm",ground[nimg]);
   else if (COLLECTION==STARE) sprintf(fichero,"images/stare/%s.pgm",ground[nimg]);
   else return;
   read_image_pgm(fichero,truth);
   if(COLLECTION==DRIVE) sprintf(fichero,"images/drive/%s.pgm",mask[nimg]);
   else if (COLLECTION==STARE) sprintf(fichero,"images/stare/%s.pgm",mask[nimg]);
   else return;
   read_image_pgm(fichero,mask_img);
   
   a00=0; a01=0; a10=0; a11=0; total=0;
   for(i=0;i<H*V;i++)
   {  if(mask_img[i]!=255) continue;
      total++;
      if     ((output[i]==1)&&(truth[i]<128)) a00++;
      else if((output[i]==1)&&(truth[i]>127)) a01++;
      else if((output[i]==0)&&(truth[i]<128)) a10++;
      else if((output[i]==0)&&(truth[i]>127)) a11++;
      else fprintf(stderr,"\n\nERROR\n\n"); }
   // sens=tp/(tp+fn), esp=tn/(tn+fp), MAA=(tp+tn)/total
   sens[nimg]=(1.0*a11)/(a11+a01); esp[nimg]=(1.0*a00)/(a00+a10);
   MAA[nimg]=(a00+a11)/(1.0*total);
   printf("Results(%s) Sens=%f, Spec=%f, MAA=%f\n",
       name[nimg],sens[nimg],esp[nimg],MAA[nimg]);
   free(mask_img);
}

void compute_averages()
{  int nimg; double t1=0, t2=0, p1=0, p2=0, p3=0;
   for(nimg=0;nimg<NIMG;nimg++)
   {  t1+=t_comp[nimg]; t2+=t_tot[nimg]; 
      p1+=sens[nimg]; p2+=esp[nimg]; p3+=MAA[nimg]; }
   printf("-----------------------------------------------\n");
   if(COLLECTION==DRIVE) printf("DRIVE"); else if(COLLECTION==STARE) printf("STARE");
   else if(COLLECTION==MESSIDOR) printf("BIG");
   printf(" GPU imag=%d stages=%d\n",NIMG,STAGES);
   printf("  Sens=%f, Spec=%f, MAA=%f\n",p1/NIMG,p2/NIMG,p3/NIMG);
   printf("  t_comp=%f, t_total=%f\n",t1/NIMG,t2/NIMG);
}
   
// ---------------------------------------------------------------------
// GPU THINGS
// ---------------------------------------------------------------------  

#define BLOCK 16 // 8 o 16, las reducciones requieren pot.2 y el filtro7 >=6
#define BLOCK2 (BLOCK+2) // para alguna funcion de etiquetado
#define BLOCKa 2   // for big images
#define BLOCKb 128
dim3 dimBlock(BLOCK,BLOCK);
dim3 dimGrid(ceilf(1.0*V/BLOCK),ceilf(1.0*H/BLOCK));
dim3 dimBlock1(BLOCK*BLOCK);
dim3 dimGrid1(ceilf(1.0*H*V/(BLOCK*BLOCK)));
// para reduccion
dim3 dimBlock2(BLOCK*BLOCK);
dim3 dimGrid2(ceilf(1.0*H*V/(BLOCK*BLOCK*BLOCK*BLOCK)));
dim3 dimBlock3(1);
dim3 dimGrid3(1);
// para el reborde
dim3 dimBlock4(BLOCK,BLOCK);
dim3 dimGrid4(ceilf(1.0*V/(BLOCK-2)),ceilf(1.0*H/(BLOCK-2)));
// para la ventana
dim3 dimGrid51(ceilf(1.0*V/BLOCK),1);
dim3 dimGrid52(1,ceilf(1.0*H/BLOCK));
// para la imagen grande
dim3 dimGrid51a(ceilf(1.0*V/BLOCKa),1);
dim3 dimGrid52a(1,ceilf(1.0*H/BLOCKa));
dim3 dimBlocka(BLOCKa,BLOCKb);
dim3 dimBlockb(BLOCKb,BLOCKa);

void cuda_init_2_0()
{  int num_devices, device, found=0;
   int max_multiprocessors = 0, max_device = 0;
   hipGetDeviceCount(&num_devices);
   for (device = 0; device < num_devices; device++) {
              hipDeviceProp_t properties;
              hipGetDeviceProperties(&properties, device);
 	      printf("Card %d: %s,",device,properties.name);
	      printf(" Capability=%d.%d, ",properties.major,properties.minor);
              if (max_multiprocessors < properties.multiProcessorCount
	           && properties.major >= 2 ) {
		      found=1;
                      max_multiprocessors = properties.multiProcessorCount;
                      max_device = device; } }
   if(found) { hipSetDevice(max_device); 
               printf("Using card num. %d\n",max_device); }
   else { fprintf(stderr,"No cards with capability >= 2.0 found\n"); exit(0); }
}

void cuda_error(const char *msg)
{   hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {   fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE); } }

__global__ void max3a(float *gi, float *go)
{ __shared__ float sdata[BLOCK*BLOCK];
  unsigned int tid=threadIdx.x;
  unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<H*V) sdata[tid]=gi[i]; else sdata[tid]=0;
  __syncthreads();
  unsigned int s;
  for(s=blockDim.x/2;s>0;s>>=1)
  {  if(tid < s) sdata[tid]=max(sdata[tid],sdata[tid+s]); 
     __syncthreads(); }
  if(tid==0) go[blockIdx.x]=sdata[0];
}

__global__ void max3b(float *gi)
{ __shared__ float sdata[BLOCK*BLOCK];
  unsigned int tid=threadIdx.x;
  unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<ceilf(1.0*H*V/(BLOCK*BLOCK))) sdata[tid]=gi[i]; else sdata[tid]=0;
  __syncthreads();
  unsigned int s;
  for(s=blockDim.x/2;s>0;s>>=1)
  {  if(tid < s) sdata[tid] = max(sdata[tid],sdata[tid+s]); 
     __syncthreads(); }
  if(tid==0) gi[blockIdx.x*BLOCK*BLOCK]=sdata[0];
}

__global__ void max3c(float *gi)
{  float tmp=0; int i, des=BLOCK*BLOCK*BLOCK*BLOCK;
   tmp=gi[0];
   for(i=1;i<ceilf(1.0*H*V/des);i++) tmp=max(tmp,gi[i*BLOCK*BLOCK]);
   gi[0]=tmp;
}

// ----------------------------------------------------------------
// UNION/FIND
// ----------------------------------------------------------------

__global__ void path_compress_gpu(int *lab)
{  unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
   int c;
   if(i<H*V) if(lab[i]<H*V) { c=i; while(lab[c]!=c) c=lab[c]; lab[i]=c; } 
}

__device__ void union_find_kernel_gpu(int *lab, int u, int v)
{  int c, d;
   c=u; while(lab[c]!=c) c=lab[c];
   d=v; while(lab[d]!=d) d=lab[d];
   if(c<d) atomicMin(&lab[d],c);
   if(d<c) atomicMin(&lab[c],d);
}

__global__ void union_find_global_gpu(int *lab, int conn)
{  int i = threadIdx.x + BLOCK*blockIdx.x;
   int j = threadIdx.y + BLOCK*blockIdx.y;

   if((i<V)&&(j<H))
   { if(lab[i*H+j]<H*V)
     { if(i>0) if(lab[(i-1)*H+j]<H*V) union_find_kernel_gpu(lab,i*H+j,(i-1)*H+j); 
       if(i<V-1) if(lab[(i+1)*H+j]<H*V) union_find_kernel_gpu(lab,i*H+j,(i+1)*H+j); 
       if(j>0) if(lab[i*H+(j-1)]<H*V) union_find_kernel_gpu(lab,i*H+j,i*H+(j-1)); 
       if(j<H-1) if(lab[i*H+(j+1)]<H*V) union_find_kernel_gpu(lab,i*H+j,i*H+(j+1)); 
       if(conn==8) {
        if((i>0)&&(j>0))if(lab[(i-1)*H+(j-1)]<H*V) union_find_kernel_gpu(lab,i*H+j,(i-1)*H+(j-1)); 
        if((i>0)&&(j<H-1))if(lab[(i-1)*H+(j+1)]<H*V) union_find_kernel_gpu(lab,i*H+j,(i-1)*H+(j+1)); 
        if((i<V-1)&&(j>0))if(lab[(i+1)*H+(j-1)]<H*V) union_find_kernel_gpu(lab,i*H+j,(i+1)*H+(j-1));
        if((i>V-1)&&(j<H-1))if(lab[(i+1)*H+(j+1)]<H*V) union_find_kernel_gpu(lab,i*H+j,(i+1)*H+(j+1)); }}}    
}

__global__ void union_find_shared_gpu(int *label, unsigned char *img, int conn)
{  unsigned int i2=threadIdx.x, j2=threadIdx.y, i=i2+1, j=j2+1, c, x, y;
   unsigned int tid=i*BLOCK2+j, tid2=i2*BLOCK+j2;
   int ig = threadIdx.x + BLOCK*blockIdx.x;
   int jg = threadIdx.y + BLOCK*blockIdx.y;
   __shared__ int lab[BLOCK2*BLOCK2]; 
   BASE F=0;
   
   if((ig<V)&&(jg<H)) { if(img[ig*H+jg]==F) lab[tid]=tid; else lab[tid]=H*V; }
   else lab[tid]=H*V;
   
   if(tid2<BLOCK2) { lab[tid2]=H*V; lab[tid2+BLOCK2*(BLOCK2-1)]=H*V;
                     lab[tid2*BLOCK2]=H*V; lab[tid2*BLOCK2+(BLOCK2-1)]=H*V; }
   __syncthreads();
   
   if(lab[tid]<H*V)
   {  if(lab[(i-1)*BLOCK2+j]<H*V) union_find_kernel_gpu(lab,tid,(i-1)*BLOCK2+j);
      if(lab[(i+1)*BLOCK2+j]<H*V) union_find_kernel_gpu(lab,tid,(i+1)*BLOCK2+j); 
      if(lab[i*BLOCK2+(j-1)]<H*V) union_find_kernel_gpu(lab,tid,i*BLOCK2+(j-1)); 
      if(lab[i*BLOCK2+(j+1)]<H*V) union_find_kernel_gpu(lab,tid,i*BLOCK2+(j+1)); 
      if(conn==8) {
        if(lab[(i-1)*BLOCK2+(j-1)]<H*V) union_find_kernel_gpu(lab,tid,(i-1)*BLOCK2+(j-1)); 
        if(lab[(i-1)*BLOCK2+(j+1)]<H*V) union_find_kernel_gpu(lab,tid,(i-1)*BLOCK2+(j+1)); 
        if(lab[(i+1)*BLOCK2+(j-1)]<H*V) union_find_kernel_gpu(lab,tid,(i+1)*BLOCK2+(j-1));
        if(lab[(i+1)*BLOCK2+(j+1)]<H*V) union_find_kernel_gpu(lab,tid,(i+1)*BLOCK2+(j+1)); } 
      __syncthreads();
 
      c=tid; while(lab[c]!=c) c=lab[c]; lab[tid]=c;  
      __syncthreads();

      x=lab[tid]/BLOCK2-1; y=(lab[tid]%BLOCK2)-1; 
      lab[tid]=(x+BLOCK*blockIdx.x)*H+(y+BLOCK*blockIdx.y); }
   __syncthreads();
      
   if((ig<V)&&(jg<H)) label[ig*H+jg]=lab[tid];
}

__global__ void update_test_gpu(int *lab, int *old, int *flag)
{  unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
   if(i<H*V) if(lab[i]!=old[i]) atomicOr(flag,1);  
}

__global__ void histogram_gpu(BASE *img, int *label, int *cuenta)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  BASE B=1;
  if(i<H*V) if(img[i]!=B) atomicAdd(&cuenta[label[i]],1);
}

__global__ void extend_histogram_gpu(BASE *img, int *label, int *cuenta)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  BASE B=1;
  if(i<H*V) if(img[i]!=B) cuenta[i]=cuenta[label[i]];
}

__global__ void remove_small_groups_gpu(BASE *img, int *cuenta, int size)
{ BASE F=0, B=1;
  unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) if((img[i]==F)&&(cuenta[i]<size)) img[i]=B;
}

void remove_noise(BASE *img, int *flag, int *label, int *cuenta, int conn, int size)
{  int stop=1;

   union_find_shared_gpu<<<dimGrid,dimBlock>>>(label,img,conn);
   union_find_global_gpu<<<dimGrid,dimBlock>>>(label,conn);
   path_compress_gpu<<<dimGrid1,dimBlock1>>>(label);
   while(stop)
   {  hipMemset(flag,0,sizeof(int));
      hipMemcpy(cuenta,label,H*V*sizeof(int),hipMemcpyDeviceToDevice);
      union_find_global_gpu<<<dimGrid,dimBlock>>>(label,conn);
      path_compress_gpu<<<dimGrid1,dimBlock1>>>(label);
      update_test_gpu<<<dimGrid1,dimBlock1>>>(label,cuenta,flag);
      hipMemcpy(&stop,flag,1*sizeof(int),hipMemcpyDeviceToHost);
   }
   hipMemset(cuenta,0,H*V*sizeof(int));
   histogram_gpu<<<dimGrid1,dimBlock1>>>(img,label,cuenta);
   extend_histogram_gpu<<<dimGrid1,dimBlock1>>>(img,label,cuenta);
   remove_small_groups_gpu<<<dimGrid1,dimBlock1>>>(img,cuenta,size);
}

__global__ void grid_cross_detection_gpu(int *label, int *aux, int SEP, int OFF1, int OFF2)
{  unsigned int i = threadIdx.x + BLOCK*blockIdx.x;
   unsigned int j = threadIdx.y + BLOCK*blockIdx.y;
   unsigned int tid=i*H+j; int c;

   if((i<V)&&(j<H)&&((i%SEP==OFF1)||(j%SEP==OFF1)))
   {  c=label[tid]; if(c<H*V) atomicOr(&aux[c],1); } 
   if((i<V)&&(j<H)&&((i%SEP==OFF2)||(j%SEP==OFF2)))
   {  c=label[tid]; if(c<H*V) atomicOr(&aux[c],2); } 
}

__global__ void extend_cross_gpu(int *label, int *aux, BASE *img)
{ BASE F=0, B=1; int c;
  unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) {  c=label[i]; if((c<H*V)&&(aux[c]==3)) img[i]=F; else img[i]=B; }
}

void remove_small_structures(BASE *img, int *flag, int *label, int *aux, int conn, int fill)
{  int stop=1;

   union_find_shared_gpu<<<dimGrid,dimBlock>>>(label,img,conn);
   union_find_global_gpu<<<dimGrid,dimBlock>>>(label,conn);
   path_compress_gpu<<<dimGrid1,dimBlock1>>>(label);
   while(stop)
   {  hipMemset(flag,0,sizeof(int));
      hipMemcpy(aux,label,H*V*sizeof(int),hipMemcpyDeviceToDevice);
      union_find_global_gpu<<<dimGrid,dimBlock>>>(label,conn);
      path_compress_gpu<<<dimGrid1,dimBlock1>>>(label);
      update_test_gpu<<<dimGrid1,dimBlock1>>>(label,aux,flag);
      hipMemcpy(&stop,flag,1*sizeof(int),hipMemcpyDeviceToHost);
   }

   hipMemset(aux,0,H*V*sizeof(int));
   grid_cross_detection_gpu<<<dimGrid,dimBlock>>>(label,aux,fill,0,fill/2);
   extend_cross_gpu<<<dimGrid1,dimBlock1>>>(label,aux,img);
}

// -------------------------------------------------------------
// FILTERS
// --------------------------------------------------------------

__global__ void erosion_shared_gpu(BASE in[H*V], BASE out[H*V])
{   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
    int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE ins[BLOCK*BLOCK];
    BASE min2;

    if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) ins[i*BLOCK+j]=in[ig*H+jg]; else ins[i*BLOCK+j]=255;
    __syncthreads(); 

    if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H))  { 
        min2=ins[i*BLOCK+j]&ins[(i-1)*BLOCK+(j-1)]&ins[(i-1)*BLOCK+j]
	    &ins[(i-1)*BLOCK+(j+1)]&ins[i*BLOCK+(j-1)]&ins[i*BLOCK+(j+1)]
	    &ins[(i+1)*BLOCK+(j-1)]&ins[(i+1)*BLOCK+j]&ins[(i+1)*BLOCK+(j+1)];
	out[ig*H+jg]=min2; }
}

__global__ void erosion_float_shared_gpu(float in[H*V], float out[H*V])
{   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
    int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ float ins[BLOCK*BLOCK];
    float min2;

    if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) ins[i*BLOCK+j]=in[ig*H+jg]; else ins[i*BLOCK+j]=99999999;
    __syncthreads(); 

    if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H))  { 
        min2=ins[i*BLOCK+j];
	min2=min(min2,ins[(i-1)*BLOCK+(j-1)]);
	min2=min(min2,ins[(i-1)*BLOCK+j]);
	min2=min(min2,ins[(i-1)*BLOCK+(j+1)]);
	min2=min(min2,ins[i*BLOCK+(j-1)] );
	min2=min(min2,ins[i*BLOCK+(j+1)]);
	min2=min(min2,ins[(i+1)*BLOCK+(j-1)]);
	min2=min(min2,ins[(i+1)*BLOCK+j]);
	min2=min(min2,ins[(i+1)*BLOCK+(j+1)]);
	out[ig*H+jg]=min2; }
}

__global__ void dilate_shared_gpu(BASE in[H*V], BASE out[H*V])
{   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
    int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE ins[BLOCK*BLOCK];
    BASE max2;

    if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) ins[i*BLOCK+j]=in[ig*H+jg]; else ins[i*BLOCK+j]=0;
    __syncthreads(); 

    if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H))  { 
        max2=ins[i*BLOCK+j]|ins[(i-1)*BLOCK+(j-1)]|ins[(i-1)*BLOCK+j]
	    |ins[(i-1)*BLOCK+(j+1)]|ins[i*BLOCK+(j-1)]|ins[i*BLOCK+(j+1)]
	    |ins[(i+1)*BLOCK+(j-1)]|ins[(i+1)*BLOCK+j]|ins[(i+1)*BLOCK+(j+1)];
	out[ig*H+jg]=max2; }
}

__global__ void f_edge_shared_gpu(BASE *A, BASE *out)
{  int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
   int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
   unsigned int i=threadIdx.x, j=threadIdx.y;
   BASE tmp, sum;
   __shared__ BASE in[BLOCK*BLOCK];

   if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) tmp=A[ig*H+jg]; else tmp=1;
   in[i*BLOCK+j]=tmp;
    __syncthreads(); 
   
   if((j>0)&&(j<BLOCK-1)&&(i>0)&&(i<BLOCK-1))
   {  sum=tmp&in[(i-1)*BLOCK+j]&in[(i+1)*BLOCK+j]&in[i*BLOCK+(j-1)]&in[i*BLOCK+(j+1)]; 
      if(sum) tmp=0; }
   if((jg<H)&&(ig<V)&&(j>0)&&(j<BLOCK-1)&&(i>0)&&(i<BLOCK-1)) 
      out[ig*H+jg]=tmp;
}

// es mas rapida en memoria global que en shared
__global__ void f_shift_global_gpu(const char dir, BASE *input, BASE *result,
     BASE boundary)
{  unsigned int i = threadIdx.x + BLOCK*blockIdx.x;
   unsigned int j = threadIdx.y + BLOCK*blockIdx.y;
   if((i<V)&&(j<H)) {
     if((dir=='S')&&(i>0)) result[i*H+j]=input[(i-1)*H+j];
     else if((dir=='W')&&(j<H-1)) result[i*H+j]=input[i*H+(j+1)];
     else if((dir=='N')&&(i<V-1)) result[i*H+j]=input[(i+1)*H+j];
     else if((dir=='E')&&(j>0))   result[i*H+j]=input[i*H+(j-1)];
     else result[i*H+j]=boundary; }
}
// vale tanto para replicar o no, para replica=1 
__global__ void filter_float_shared_gpu(float A[H*V], float aux[H*V], int replica)
{   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
    int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    float B[]={0.10,0.15,0.10, 0.15,0.00,0.15, 0.10, 0.15, 0.10};
    float tmp;
    __shared__ float in[BLOCK*BLOCK];

    if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) in[i*BLOCK+j]=A[ig*H+jg]; else in[i*BLOCK+j]=0;
    __syncthreads(); 

    if(replica==1)
    {  if((ig<0)&&(jg<0)) in[i*BLOCK+j]=in[(i+1)*BLOCK+(j+1)];
       else if((ig<0)&&(jg==H)) in[i*BLOCK+j]=in[(i+1)*BLOCK+(j-1)];
       else if((ig==V)&&(jg<0)) in[i*BLOCK+j]=in[(i-1)*BLOCK+(j+1)];
       else if((ig==V)&&(jg==H)) in[i*BLOCK+j]=in[(i-1)*BLOCK+(j-1)];
       else if(ig<0) in[i*BLOCK+j]=in[(i+1)*BLOCK+j];
       else if(ig==V) in[i*BLOCK+j]=in[(i-1)*BLOCK+j];
       else if(jg<0) in[i*BLOCK+j]=in[i*BLOCK+(j+1)];
       else if(jg==H) in[i*BLOCK+j]=in[i*BLOCK+(j-1)];
       __syncthreads(); }

    if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H))  { 
       tmp=B[0]*in[(i-1)*BLOCK+(j-1)]+B[1]*in[(i-1)*BLOCK+j]+B[2]*in[(i-1)*BLOCK+(j+1)]
          +B[3]*in[i*BLOCK+(j-1)]    +B[4]*in[i*BLOCK+j]    +B[5]*in[i*BLOCK+(j+1)]
	  +B[6]*in[(i+1)*BLOCK+(j-1)]+B[7]*in[(i+1)*BLOCK+j]+B[8]*in[(i+1)*BLOCK+(j+1)]; 
       aux[ig*H+jg]=tmp; }
}

__global__ void remove_outside_FOV_gpu(BASE *f, BASE *tmp, int umbral)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) if(tmp[i]<umbral) f[i]=1; //25
}

__global__ void sobel_edge_detect_lineal_gpu(BASE *in, BASE *out) 
{  unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
   int l, fil, col, ns=3; int gx, gy, g;
   int sobx[]={-1,0,1,-2,0,2,-1,0,1}, soby[]={1,2,1,0,0,0,-1,-2,-1};

   if(i<H*V) {
   gx=0; gy=0; for(l=0;l<ns*ns;l++)
   {  fil=(i/H)-((ns-1)/2)+(l/ns); col=(i%H)-((ns-1)/2)+(l%ns);
      if(fil<0)fil=0; else if(fil>=V) fil=V-1;
      if(col<0)col=0; else if(col>=H) col=H-1;
      gx+=sobx[l]*in[fil*H+col]; gy+=soby[l]*in[fil*H+col]; }
      g=gx*gx+gy*gy;
      if(g<SOBEL_UMBRAL) out[i]=0; else out[i]=1;
   // stagesa final   
   out[i] = in[i] & (!out[i]); }
}

__global__ void sobel_edge_detect_shared_gpu(BASE *A, BASE *out)
{   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
    int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    int gx, gy, g; BASE tmp1,tmp2;
    int BX[]={-1,0,1,-2,0,2,-1,0,1}, BY[]={1,2,1,0,0,0,-1,-2,-1};
    __shared__ int in[BLOCK*BLOCK];
     
    if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) tmp1=A[ig*H+jg]; else tmp1=0;
    in[i*BLOCK+j]=tmp1;
    __syncthreads(); 
    if((ig<0)&&(jg<0)) in[i*BLOCK+j]=in[(i+1)*BLOCK+(j+1)];
    else if((ig<0)&&(jg==H)) in[i*BLOCK+j]=in[(i+1)*BLOCK+(j-1)];
    else if((ig==V)&&(jg<0)) in[i*BLOCK+j]=in[(i-1)*BLOCK+(j+1)];
    else if((ig==V)&&(jg==H)) in[i*BLOCK+j]=in[(i-1)*BLOCK+(j-1)];
    else if(ig<0) in[i*BLOCK+j]=in[(i+1)*BLOCK+j];
    else if(ig==V) in[i*BLOCK+j]=in[(i-1)*BLOCK+j];
    else if(jg<0) in[i*BLOCK+j]=in[i*BLOCK+(j+1)];
    else if(jg==H) in[i*BLOCK+j]=in[i*BLOCK+(j-1)];    
    __syncthreads(); 
   
    if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H))  { 
       gx=BX[0]*in[(i-1)*BLOCK+(j-1)]+BX[1]*in[(i-1)*BLOCK+j]+BX[2]*in[(i-1)*BLOCK+(j+1)]
         +BX[3]*in[i*BLOCK+(j-1)]    +BX[4]*in[i*BLOCK+j]    +BX[5]*in[i*BLOCK+(j+1)]
	 +BX[6]*in[(i+1)*BLOCK+(j-1)]+BX[7]*in[(i+1)*BLOCK+j]+BX[8]*in[(i+1)*BLOCK+(j+1)];
       gy=BY[0]*in[(i-1)*BLOCK+(j-1)]+BY[1]*in[(i-1)*BLOCK+j]+BY[2]*in[(i-1)*BLOCK+(j+1)]
         +BY[3]*in[i*BLOCK+(j-1)]    +BY[4]*in[i*BLOCK+j]    +BY[5]*in[i*BLOCK+(j+1)]
	 +BY[6]*in[(i+1)*BLOCK+(j-1)]+BY[7]*in[(i+1)*BLOCK+j]+BY[8]*in[(i+1)*BLOCK+(j+1)];
       g=gx*gx+gy*gy;	 
       __syncthreads(); 
       if(g<SOBEL_UMBRAL) tmp2=0; else tmp2=1; 
       // esto no tiene que ver con la deteccion de edges
       out[ig*H+jg]= tmp1 & (!tmp2); }
}

__global__ void erosion_st15_filas_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x;
    int jg = threadIdx.y + BLOCK*blockIdx.y-BLOCK/2;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[BLOCK][2*BLOCK];
    int k, l; BASE min2;

    in[i][j+BLOCK]=((ig<V)&&(jg>=0)&&(jg<H))?A[ig*H+jg]:255;
    __syncthreads();

    for(k=0;k<ceilf(1.0*H/BLOCK);k++)
    {  jg=jg+BLOCK;
       in[i][j]=in[i][j+BLOCK];
       in[i][j+BLOCK]=((ig<V)&&(jg<H))?A[ig*H+jg]:255;
       __syncthreads();
       if((ig<V)&&(jg-BLOCK/2<H)) { min2=in[i][j+BLOCK/2];
          for(l=-7;l<=7;l++) min2=min(min2,in[i][j+BLOCK/2+l]);
       out[ig*H+jg-BLOCK/2]= (BASE) min2; } 
       __syncthreads(); }
}

__global__ void erosion_st15_cols_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x-BLOCK/2;
    int jg = threadIdx.y + BLOCK*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[2*BLOCK][BLOCK];
    int k, l; BASE min2;

    in[i+BLOCK][j]=((ig>=0)&&(ig<V)&&(jg<H))?A[ig*H+jg]:255;
    __syncthreads();

    for(k=0;k<ceilf(1.0*V/BLOCK);k++)
    {  ig=ig+BLOCK;
       in[i][j]=in[i+BLOCK][j];
       in[i+BLOCK][j]=((ig<V)&&(jg<H))?A[ig*H+jg]:255;
       __syncthreads();
       if((ig-BLOCK/2<V)&&(jg<H)) { min2=in[i+BLOCK/2][j];
          for(l=-7;l<=7;l++) min2=min(min2,in[i+BLOCK/2+l][j]);
       out[(ig-BLOCK/2)*H+jg]= (BASE) min2; } 
       __syncthreads(); }
}


__global__ void filter_st17_filas_win_gpu(BASE A[H*V], float out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x;
    int jg = threadIdx.y + BLOCK*blockIdx.y-BLOCK/2;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[BLOCK][2*BLOCK];
    int k, l, a, b; float tmp;

    a=ig; if(a>=V) a=V-1; b=jg; if(b<0) b=0; else if(b>=H) b=H-1; 
    in[i][j+BLOCK]=A[a*H+b];
    __syncthreads();

    for(k=0;k<ceilf(1.0*H/BLOCK);k++)
    {  jg=jg+BLOCK; b=jg; if(b>=H) b=H-1;
       in[i][j]=in[i][j+BLOCK];
       in[i][j+BLOCK]=A[a*H+b];
       __syncthreads();
       if((ig<V)&&(jg-BLOCK/2<H)) 
       {  tmp=0; 
          for(l=-8;l<=8;l++) tmp+=in[i][j+BLOCK/2+l];
          out[ig*H+jg-BLOCK/2]= (float) tmp; } 
       __syncthreads(); }
}

__global__ void filter_st17_cols_win_gpu(float A[H*V], float out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x-BLOCK/2;
    int jg = threadIdx.y + BLOCK*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ float in[2*BLOCK][BLOCK];
    int k, l, a, b; float tmp=0;

    b=jg; if(b>=H) b=H-1; a=ig; if(a<0) a=0; else if(a>=V) a=V-1; 
    in[i+BLOCK][j]=A[a*H+b];
    __syncthreads();

    for(k=0;k<ceilf(1.0*V/BLOCK);k++)
    {  ig=ig+BLOCK; a=ig; if(a>=V) a=V-1; 
       in[i][j]=in[i+BLOCK][j];
       in[i+BLOCK][j]=A[a*H+b];
       __syncthreads();
       if((ig-BLOCK/2<V)&&(jg<H)) 
       {  tmp=0;
          for(l=-8;l<=8;l++) tmp+=in[i+BLOCK/2+l][j];
          out[(ig-BLOCK/2)*H+jg]= tmp/289.0; } 
       __syncthreads(); }
}

__global__ void kernel_resta(BASE *im, float *imblur)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) { imblur[i]=im[i]-imblur[i]; } // 
}

__global__ void kernel_thresholding(float *imblur, BASE *tmp1, float factor)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) { tmp1[i]=(imblur[i]+factor>0); }
}

void diffusion_and_binarization(BASE *tmp1, BASE *imt, float *imblur, float *aux1, float factor1)
{  filter_st17_filas_win_gpu<<<dimGrid51,dimBlock>>>(tmp1,aux1);
   filter_st17_cols_win_gpu<<<dimGrid52,dimBlock>>>(aux1,imblur);
   kernel_resta<<<dimGrid1,dimBlock1>>>(tmp1,imblur);  
   kernel_thresholding<<<dimGrid1,dimBlock1>>>(imblur,imt,factor1); 
}

void postproc_big_images(BASE *output, BASE *tmp1)
{  erosion_shared_gpu<<<dimGrid4,dimBlock4>>>(output,tmp1);
   erosion_shared_gpu<<<dimGrid4,dimBlock4>>>(tmp1,output);
   erosion_shared_gpu<<<dimGrid4,dimBlock4>>>(output,tmp1);
   erosion_shared_gpu<<<dimGrid4,dimBlock4>>>(tmp1,output);
   if(H>4000)
   {  erosion_shared_gpu<<<dimGrid4,dimBlock4>>>(output,tmp1);
      dilate_shared_gpu<<<dimGrid4,dimBlock4>>>(tmp1,output); }
   dilate_shared_gpu<<<dimGrid4,dimBlock4>>>(output,tmp1);
   dilate_shared_gpu<<<dimGrid4,dimBlock4>>>(tmp1,output);
   dilate_shared_gpu<<<dimGrid4,dimBlock4>>>(output,tmp1);
   dilate_shared_gpu<<<dimGrid4,dimBlock4>>>(tmp1,output);
}

//-------------------------------------------------------------
// TOP-HAT
//-------------------------------------------------------------

__global__ void dilate_st15_filas_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x;
    int jg = threadIdx.y + BLOCK*blockIdx.y-BLOCK/2;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[BLOCK][2*BLOCK];
    int k, l; BASE max2;

    in[i][j+BLOCK]=((ig<V)&&(jg>=0)&&(jg<H))?A[ig*H+jg]:0;
    __syncthreads();

    for(k=0;k<ceilf(1.0*H/BLOCK);k++)
    {  jg=jg+BLOCK;
       in[i][j]=in[i][j+BLOCK];
       in[i][j+BLOCK]=((ig<V)&&(jg<H))?A[ig*H+jg]:0;
       __syncthreads();
       if((ig<V)&&(jg-BLOCK/2<H)) { max2=in[i][j+BLOCK/2];
          for(l=-7;l<=7;l++) max2=max(max2,in[i][j+BLOCK/2+l]);
       out[ig*H+jg-BLOCK/2]= (BASE) max2; } 
       __syncthreads(); }
}

__global__ void dilate_st15_cols_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x-BLOCK/2;
    int jg = threadIdx.y + BLOCK*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[2*BLOCK][BLOCK];
    int k, l; BASE max2;

    in[i+BLOCK][j]=((ig>=0)&&(ig<V)&&(jg<H))?A[ig*H+jg]:0;
    __syncthreads();

    for(k=0;k<ceilf(1.0*V/BLOCK);k++)
    {  ig=ig+BLOCK;
       in[i][j]=in[i+BLOCK][j];
       in[i+BLOCK][j]=((ig<V)&&(jg<H))?A[ig*H+jg]:0;
       __syncthreads();
       if((ig-BLOCK/2<V)&&(jg<H)) { max2=in[i+BLOCK/2][j];
          for(l=-7;l<=7;l++) max2=max(max2,in[i+BLOCK/2+l][j]);
       out[(ig-BLOCK/2)*H+jg]= (BASE) max2; } 
       __syncthreads(); }
}

__global__ void erosion_st64_filas_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCKa*blockIdx.x;
    int jg = threadIdx.y + BLOCKb*blockIdx.y-BLOCKb/2;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[BLOCKa][2*BLOCKb];
    int k, l; BASE min2;

    in[i][j+BLOCKb]=((ig<V)&&(jg>=0)&&(jg<H))?A[ig*H+jg]:255;
    __syncthreads();

    for(k=0;k<ceilf(1.0*H/BLOCKb);k++)
    {  jg=jg+BLOCKb;
       in[i][j]=in[i][j+BLOCKb];
       in[i][j+BLOCKb]=((ig<V)&&(jg<H))?A[ig*H+jg]:255;
       __syncthreads();
       if((ig<V)&&(jg-BLOCKb/2<H)) { min2=in[i][j+BLOCKb/2];
          for(l=-64;l<=64;l++) min2=min(min2,in[i][j+BLOCKb/2+l]);
       out[ig*H+jg-BLOCKb/2]= (BASE) min2; } 
       __syncthreads(); }
}

__global__ void erosion_st64_cols_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCKb*blockIdx.x-BLOCKb/2;
    int jg = threadIdx.y + BLOCKa*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[2*BLOCKb][BLOCKa];
    int k, l; BASE min2;

    in[i+BLOCKb][j]=((ig>=0)&&(ig<V)&&(jg<H))?A[ig*H+jg]:255;
    __syncthreads();

    for(k=0;k<ceilf(1.0*V/BLOCKb);k++)
    {  ig=ig+BLOCKb;
       in[i][j]=in[i+BLOCKb][j];
       in[i+BLOCKb][j]=((ig<V)&&(jg<H))?A[ig*H+jg]:255;
       __syncthreads();
       if((ig-BLOCKb/2<V)&&(jg<H)) { min2=in[i+BLOCKb/2][j];
          for(l=-64;l<=64;l++) min2=min(min2,in[i+BLOCKb/2+l][j]);
       out[(ig-BLOCKb/2)*H+jg]= (BASE) min2; } 
       __syncthreads(); }
}

__global__ void dilate_st64_filas_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCKa*blockIdx.x;
    int jg = threadIdx.y + BLOCKb*blockIdx.y-BLOCKb/2;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[BLOCKa][2*BLOCKb];
    int k, l; BASE max2;

    in[i][j+BLOCKb]=((ig<V)&&(jg>=0)&&(jg<H))?A[ig*H+jg]:0;
    __syncthreads();

    for(k=0;k<ceilf(1.0*H/BLOCKb);k++)
    {  jg=jg+BLOCKb;
       in[i][j]=in[i][j+BLOCKb];
       in[i][j+BLOCKb]=((ig<V)&&(jg<H))?A[ig*H+jg]:0;
       __syncthreads();
       if((ig<V)&&(jg-BLOCKb/2<H)) { max2=in[i][j+BLOCKb/2];
          for(l=-64;l<=64;l++) max2=max(max2,in[i][j+BLOCKb/2+l]);
       out[ig*H+jg-BLOCKb/2]= (BASE) max2; } 
       __syncthreads(); }
}

__global__ void dilate_st64_cols_win_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCKb*blockIdx.x-BLOCKb/2;
    int jg = threadIdx.y + BLOCKa*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y;
    __shared__ BASE in[2*BLOCKb][BLOCKa];
    int k, l; BASE max2;

    in[i+BLOCKb][j]=((ig>=0)&&(ig<V)&&(jg<H))?A[ig*H+jg]:0;
    __syncthreads();

    for(k=0;k<ceilf(1.0*V/BLOCKb);k++)
    {  ig=ig+BLOCKb;
       in[i][j]=in[i+BLOCKb][j];
       in[i+BLOCKb][j]=((ig<V)&&(jg<H))?A[ig*H+jg]:0;
       __syncthreads();
       if((ig-BLOCKb/2<V)&&(jg<H)) { max2=in[i+BLOCKb/2][j];
          for(l=-64;l<=64;l++) max2=max(max2,in[i+BLOCKb/2+l][j]);
       out[(ig-BLOCKb/2)*H+jg]= (BASE) max2; } 
       __syncthreads(); }
}

__global__ void erosion_disk_st15_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x;
    int jg = threadIdx.y + BLOCK*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y; int a,b;
    __shared__ BASE in[2*BLOCK][2*BLOCK];
    BASE min2;

    a=ig-BLOCK/2;if(a<0)a=0;else if(a>=V) a=V-1;
    b=jg-BLOCK/2;if(b<0)b=0;else if(b>=H) b=H-1;
    in[i][j]=A[a*H+b];
    a=ig-BLOCK/2;if(a<0)a=0;else if(a>=V) a=V-1;
    b=jg+BLOCK/2;if(b>=H)b=H-1;
    in[i][j+BLOCK]=A[a*H+b];
    a=ig+BLOCK/2;if(a>=V)a=V-1;
    b=jg-BLOCK/2;if(b<0)b=0;else if(b>=H)b=H-1;
    in[i+BLOCK][j]=A[a*H+b];
    a=ig+BLOCK/2;if(a>=V)a=V-1;
    b=jg+BLOCK/2;if(b>=H)b=H-1;
    in[i+BLOCK][j+BLOCK]=A[a*H+b];
    __syncthreads();

    if((ig<V)&&(jg<H)) 
    {  a=i+BLOCK/2; b=j+BLOCK/2;
       min2=in[a][b];
       min2=min(min2,in[a-7][b-3]);
       min2=min(min2,in[a-7][b-2]);
       min2=min(min2,in[a-7][b-1]);
       min2=min(min2,in[a-7][b+0]);
       min2=min(min2,in[a-7][b+1]);
       min2=min(min2,in[a-7][b+2]);
       min2=min(min2,in[a-7][b+3]);
       min2=min(min2,in[a-6][b-4]);
       min2=min(min2,in[a-6][b-3]);
       min2=min(min2,in[a-6][b-2]);
       min2=min(min2,in[a-6][b-1]);
       min2=min(min2,in[a-6][b+0]);
       min2=min(min2,in[a-6][b+1]);
       min2=min(min2,in[a-6][b+2]);
       min2=min(min2,in[a-6][b+3]);
       min2=min(min2,in[a-6][b+4]);
       min2=min(min2,in[a-5][b-5]);
       min2=min(min2,in[a-5][b-4]);
       min2=min(min2,in[a-5][b-3]);
       min2=min(min2,in[a-5][b-2]);
       min2=min(min2,in[a-5][b-1]);
       min2=min(min2,in[a-5][b+0]);
       min2=min(min2,in[a-5][b+1]);
       min2=min(min2,in[a-5][b+2]);
       min2=min(min2,in[a-5][b+3]);
       min2=min(min2,in[a-5][b+4]);
       min2=min(min2,in[a-5][b+5]);
       min2=min(min2,in[a-4][b-6]);
       min2=min(min2,in[a-4][b-5]);
       min2=min(min2,in[a-4][b-4]);
       min2=min(min2,in[a-4][b-3]);
       min2=min(min2,in[a-4][b-2]);
       min2=min(min2,in[a-4][b-1]);
       min2=min(min2,in[a-4][b+0]);
       min2=min(min2,in[a-4][b+1]);
       min2=min(min2,in[a-4][b+2]);
       min2=min(min2,in[a-4][b+3]);
       min2=min(min2,in[a-4][b+4]);
       min2=min(min2,in[a-4][b+5]);
       min2=min(min2,in[a-4][b+6]);
       min2=min(min2,in[a-3][b-7]);
       min2=min(min2,in[a-3][b-6]);
       min2=min(min2,in[a-3][b-5]);
       min2=min(min2,in[a-3][b-4]);
       min2=min(min2,in[a-3][b-3]);
       min2=min(min2,in[a-3][b-2]);
       min2=min(min2,in[a-3][b-1]);
       min2=min(min2,in[a-3][b+0]);
       min2=min(min2,in[a-3][b+1]);
       min2=min(min2,in[a-3][b+2]);
       min2=min(min2,in[a-3][b+3]);
       min2=min(min2,in[a-3][b+4]);
       min2=min(min2,in[a-3][b+5]);
       min2=min(min2,in[a-3][b+6]);
       min2=min(min2,in[a-3][b+7]);
       min2=min(min2,in[a-2][b-7]);
       min2=min(min2,in[a-2][b-6]);
       min2=min(min2,in[a-2][b-5]);
       min2=min(min2,in[a-2][b-4]);
       min2=min(min2,in[a-2][b-3]);
       min2=min(min2,in[a-2][b-2]);
       min2=min(min2,in[a-2][b-1]);
       min2=min(min2,in[a-2][b+0]);
       min2=min(min2,in[a-2][b+1]);
       min2=min(min2,in[a-2][b+2]);
       min2=min(min2,in[a-2][b+3]);
       min2=min(min2,in[a-2][b+4]);
       min2=min(min2,in[a-2][b+5]);
       min2=min(min2,in[a-2][b+6]);
       min2=min(min2,in[a-2][b+7]);
       min2=min(min2,in[a-1][b-7]);
       min2=min(min2,in[a-1][b-6]);
       min2=min(min2,in[a-1][b-5]);
       min2=min(min2,in[a-1][b-4]);
       min2=min(min2,in[a-1][b-3]);
       min2=min(min2,in[a-1][b-2]);
       min2=min(min2,in[a-1][b-1]);
       min2=min(min2,in[a-1][b+0]);
       min2=min(min2,in[a-1][b+1]);
       min2=min(min2,in[a-1][b+2]);
       min2=min(min2,in[a-1][b+3]);
       min2=min(min2,in[a-1][b+4]);
       min2=min(min2,in[a-1][b+5]);
       min2=min(min2,in[a-1][b+6]);
       min2=min(min2,in[a-1][b+7]);
       min2=min(min2,in[a+0][b-7]);
       min2=min(min2,in[a+0][b-6]);
       min2=min(min2,in[a+0][b-5]);
       min2=min(min2,in[a+0][b-4]);
       min2=min(min2,in[a+0][b-3]);
       min2=min(min2,in[a+0][b-2]);
       min2=min(min2,in[a+0][b-1]);
       min2=min(min2,in[a+0][b+1]);
       min2=min(min2,in[a+0][b+2]);
       min2=min(min2,in[a+0][b+3]);
       min2=min(min2,in[a+0][b+4]);
       min2=min(min2,in[a+0][b+5]);
       min2=min(min2,in[a+0][b+6]);
       min2=min(min2,in[a+0][b+7]);
       min2=min(min2,in[a+1][b-7]);
       min2=min(min2,in[a+1][b-6]);
       min2=min(min2,in[a+1][b-5]);
       min2=min(min2,in[a+1][b-4]);
       min2=min(min2,in[a+1][b-3]);
       min2=min(min2,in[a+1][b-2]);
       min2=min(min2,in[a+1][b-1]);
       min2=min(min2,in[a+1][b+0]);
       min2=min(min2,in[a+1][b+1]);
       min2=min(min2,in[a+1][b+2]);
       min2=min(min2,in[a+1][b+3]);
       min2=min(min2,in[a+1][b+4]);
       min2=min(min2,in[a+1][b+5]);
       min2=min(min2,in[a+1][b+6]);
       min2=min(min2,in[a+1][b+7]);
       min2=min(min2,in[a+2][b-7]);
       min2=min(min2,in[a+2][b-6]);
       min2=min(min2,in[a+2][b-5]);
       min2=min(min2,in[a+2][b-4]);
       min2=min(min2,in[a+2][b-3]);
       min2=min(min2,in[a+2][b-2]);
       min2=min(min2,in[a+2][b-1]);
       min2=min(min2,in[a+2][b+0]);
       min2=min(min2,in[a+2][b+1]);
       min2=min(min2,in[a+2][b+2]);
       min2=min(min2,in[a+2][b+3]);
       min2=min(min2,in[a+2][b+4]);
       min2=min(min2,in[a+2][b+5]);
       min2=min(min2,in[a+2][b+6]);
       min2=min(min2,in[a+2][b+7]);
       min2=min(min2,in[a+3][b-7]);
       min2=min(min2,in[a+3][b-6]);
       min2=min(min2,in[a+3][b-5]);
       min2=min(min2,in[a+3][b-4]);
       min2=min(min2,in[a+3][b-3]);
       min2=min(min2,in[a+3][b-2]);
       min2=min(min2,in[a+3][b-1]);
       min2=min(min2,in[a+3][b+0]);
       min2=min(min2,in[a+3][b+1]);
       min2=min(min2,in[a+3][b+2]);
       min2=min(min2,in[a+3][b+3]);
       min2=min(min2,in[a+3][b+4]);
       min2=min(min2,in[a+3][b+5]);
       min2=min(min2,in[a+3][b+6]);
       min2=min(min2,in[a+3][b+7]);
       min2=min(min2,in[a+4][b-6]);
       min2=min(min2,in[a+4][b-5]);
       min2=min(min2,in[a+4][b-4]);
       min2=min(min2,in[a+4][b-3]);
       min2=min(min2,in[a+4][b-2]);
       min2=min(min2,in[a+4][b-1]);
       min2=min(min2,in[a+4][b+0]);
       min2=min(min2,in[a+4][b+1]);
       min2=min(min2,in[a+4][b+2]);
       min2=min(min2,in[a+4][b+3]);
       min2=min(min2,in[a+4][b+4]);
       min2=min(min2,in[a+4][b+5]);
       min2=min(min2,in[a+4][b+6]);
       min2=min(min2,in[a+5][b-5]);
       min2=min(min2,in[a+5][b-4]);
       min2=min(min2,in[a+5][b-3]);
       min2=min(min2,in[a+5][b-2]);
       min2=min(min2,in[a+5][b-1]);
       min2=min(min2,in[a+5][b+0]);
       min2=min(min2,in[a+5][b+1]);
       min2=min(min2,in[a+5][b+2]);
       min2=min(min2,in[a+5][b+3]);
       min2=min(min2,in[a+5][b+4]);
       min2=min(min2,in[a+5][b+5]);
       min2=min(min2,in[a+6][b-4]);
       min2=min(min2,in[a+6][b-3]);
       min2=min(min2,in[a+6][b-2]);
       min2=min(min2,in[a+6][b-1]);
       min2=min(min2,in[a+6][b+0]);
       min2=min(min2,in[a+6][b+1]);
       min2=min(min2,in[a+6][b+2]);
       min2=min(min2,in[a+6][b+3]);
       min2=min(min2,in[a+6][b+4]);
       min2=min(min2,in[a+7][b-3]);
       min2=min(min2,in[a+7][b-2]);
       min2=min(min2,in[a+7][b-1]);
       min2=min(min2,in[a+7][b+0]);
       min2=min(min2,in[a+7][b+1]);
       min2=min(min2,in[a+7][b+2]);
       min2=min(min2,in[a+7][b+3]);    
       out[ig*H+jg]=min2; }
}

__global__ void dilate_disk_st15_gpu(BASE A[H*V], BASE out[H*V])
{   int ig = threadIdx.x + BLOCK*blockIdx.x;
    int jg = threadIdx.y + BLOCK*blockIdx.y;
    unsigned int i=threadIdx.x, j=threadIdx.y; int a,b; 
    __shared__ BASE in[2*BLOCK][2*BLOCK];
    BASE max2;

    a=ig-BLOCK/2;if(a<0)a=0;else if(a>=V) a=V-1;
    b=jg-BLOCK/2;if(b<0)b=0;else if(b>=H) b=H-1;
    in[i][j]=A[a*H+b];
    a=ig-BLOCK/2;if(a<0)a=0;else if(a>=V) a=V-1;
    b=jg+BLOCK/2;if(b>=H)b=H-1;
    in[i][j+BLOCK]=A[a*H+b];
    a=ig+BLOCK/2;if(a>=V)a=V-1;
    b=jg-BLOCK/2;if(b<0)b=0;else if(b>=H)b=H-1;
    in[i+BLOCK][j]=A[a*H+b];
    a=ig+BLOCK/2;if(a>=V)a=V-1;
    b=jg+BLOCK/2;if(b>=H)b=H-1;
    in[i+BLOCK][j+BLOCK]=A[a*H+b];
    __syncthreads();

    if((ig<V)&&(jg<H)) 
    {  a=i+BLOCK/2; b=j+BLOCK/2;
       max2=in[a][b];
       max2=max(max2,in[a-7][b-3]);
       max2=max(max2,in[a-7][b-2]);
       max2=max(max2,in[a-7][b-1]);
       max2=max(max2,in[a-7][b+0]);
       max2=max(max2,in[a-7][b+1]);
       max2=max(max2,in[a-7][b+2]);
       max2=max(max2,in[a-7][b+3]);
       max2=max(max2,in[a-6][b-4]);
       max2=max(max2,in[a-6][b-3]);
       max2=max(max2,in[a-6][b-2]);
       max2=max(max2,in[a-6][b-1]);
       max2=max(max2,in[a-6][b+0]);
       max2=max(max2,in[a-6][b+1]);
       max2=max(max2,in[a-6][b+2]);
       max2=max(max2,in[a-6][b+3]);
       max2=max(max2,in[a-6][b+4]);
       max2=max(max2,in[a-5][b-5]);
       max2=max(max2,in[a-5][b-4]);
       max2=max(max2,in[a-5][b-3]);
       max2=max(max2,in[a-5][b-2]);
       max2=max(max2,in[a-5][b-1]);
       max2=max(max2,in[a-5][b+0]);
       max2=max(max2,in[a-5][b+1]);
       max2=max(max2,in[a-5][b+2]);
       max2=max(max2,in[a-5][b+3]);
       max2=max(max2,in[a-5][b+4]);
       max2=max(max2,in[a-5][b+5]);
       max2=max(max2,in[a-4][b-6]);
       max2=max(max2,in[a-4][b-5]);
       max2=max(max2,in[a-4][b-4]);
       max2=max(max2,in[a-4][b-3]);
       max2=max(max2,in[a-4][b-2]);
       max2=max(max2,in[a-4][b-1]);
       max2=max(max2,in[a-4][b+0]);
       max2=max(max2,in[a-4][b+1]);
       max2=max(max2,in[a-4][b+2]);
       max2=max(max2,in[a-4][b+3]);
       max2=max(max2,in[a-4][b+4]);
       max2=max(max2,in[a-4][b+5]);
       max2=max(max2,in[a-4][b+6]);
       max2=max(max2,in[a-3][b-7]);
       max2=max(max2,in[a-3][b-6]);
       max2=max(max2,in[a-3][b-5]);
       max2=max(max2,in[a-3][b-4]);
       max2=max(max2,in[a-3][b-3]);
       max2=max(max2,in[a-3][b-2]);
       max2=max(max2,in[a-3][b-1]);
       max2=max(max2,in[a-3][b+0]);
       max2=max(max2,in[a-3][b+1]);
       max2=max(max2,in[a-3][b+2]);
       max2=max(max2,in[a-3][b+3]);
       max2=max(max2,in[a-3][b+4]);
       max2=max(max2,in[a-3][b+5]);
       max2=max(max2,in[a-3][b+6]);
       max2=max(max2,in[a-3][b+7]);
       max2=max(max2,in[a-2][b-7]);
       max2=max(max2,in[a-2][b-6]);
       max2=max(max2,in[a-2][b-5]);
       max2=max(max2,in[a-2][b-4]);
       max2=max(max2,in[a-2][b-3]);
       max2=max(max2,in[a-2][b-2]);
       max2=max(max2,in[a-2][b-1]);
       max2=max(max2,in[a-2][b+0]);
       max2=max(max2,in[a-2][b+1]);
       max2=max(max2,in[a-2][b+2]);
       max2=max(max2,in[a-2][b+3]);
       max2=max(max2,in[a-2][b+4]);
       max2=max(max2,in[a-2][b+5]);
       max2=max(max2,in[a-2][b+6]);
       max2=max(max2,in[a-2][b+7]);
       max2=max(max2,in[a-1][b-7]);
       max2=max(max2,in[a-1][b-6]);
       max2=max(max2,in[a-1][b-5]);
       max2=max(max2,in[a-1][b-4]);
       max2=max(max2,in[a-1][b-3]);
       max2=max(max2,in[a-1][b-2]);
       max2=max(max2,in[a-1][b-1]);
       max2=max(max2,in[a-1][b+0]);
       max2=max(max2,in[a-1][b+1]);
       max2=max(max2,in[a-1][b+2]);
       max2=max(max2,in[a-1][b+3]);
       max2=max(max2,in[a-1][b+4]);
       max2=max(max2,in[a-1][b+5]);
       max2=max(max2,in[a-1][b+6]);
       max2=max(max2,in[a-1][b+7]);
       max2=max(max2,in[a+0][b-7]);
       max2=max(max2,in[a+0][b-6]);
       max2=max(max2,in[a+0][b-5]);
       max2=max(max2,in[a+0][b-4]);
       max2=max(max2,in[a+0][b-3]);
       max2=max(max2,in[a+0][b-2]);
       max2=max(max2,in[a+0][b-1]);
       max2=max(max2,in[a+0][b+1]);
       max2=max(max2,in[a+0][b+2]);
       max2=max(max2,in[a+0][b+3]);
       max2=max(max2,in[a+0][b+4]);
       max2=max(max2,in[a+0][b+5]);
       max2=max(max2,in[a+0][b+6]);
       max2=max(max2,in[a+0][b+7]);
       max2=max(max2,in[a+1][b-7]);
       max2=max(max2,in[a+1][b-6]);
       max2=max(max2,in[a+1][b-5]);
       max2=max(max2,in[a+1][b-4]);
       max2=max(max2,in[a+1][b-3]);
       max2=max(max2,in[a+1][b-2]);
       max2=max(max2,in[a+1][b-1]);
       max2=max(max2,in[a+1][b+0]);
       max2=max(max2,in[a+1][b+1]);
       max2=max(max2,in[a+1][b+2]);
       max2=max(max2,in[a+1][b+3]);
       max2=max(max2,in[a+1][b+4]);
       max2=max(max2,in[a+1][b+5]);
       max2=max(max2,in[a+1][b+6]);
       max2=max(max2,in[a+1][b+7]);
       max2=max(max2,in[a+2][b-7]);
       max2=max(max2,in[a+2][b-6]);
       max2=max(max2,in[a+2][b-5]);
       max2=max(max2,in[a+2][b-4]);
       max2=max(max2,in[a+2][b-3]);
       max2=max(max2,in[a+2][b-2]);
       max2=max(max2,in[a+2][b-1]);
       max2=max(max2,in[a+2][b+0]);
       max2=max(max2,in[a+2][b+1]);
       max2=max(max2,in[a+2][b+2]);
       max2=max(max2,in[a+2][b+3]);
       max2=max(max2,in[a+2][b+4]);
       max2=max(max2,in[a+2][b+5]);
       max2=max(max2,in[a+2][b+6]);
       max2=max(max2,in[a+2][b+7]);
       max2=max(max2,in[a+3][b-7]);
       max2=max(max2,in[a+3][b-6]);
       max2=max(max2,in[a+3][b-5]);
       max2=max(max2,in[a+3][b-4]);
       max2=max(max2,in[a+3][b-3]);
       max2=max(max2,in[a+3][b-2]);
       max2=max(max2,in[a+3][b-1]);
       max2=max(max2,in[a+3][b+0]);
       max2=max(max2,in[a+3][b+1]);
       max2=max(max2,in[a+3][b+2]);
       max2=max(max2,in[a+3][b+3]);
       max2=max(max2,in[a+3][b+4]);
       max2=max(max2,in[a+3][b+5]);
       max2=max(max2,in[a+3][b+6]);
       max2=max(max2,in[a+3][b+7]);
       max2=max(max2,in[a+4][b-6]);
       max2=max(max2,in[a+4][b-5]);
       max2=max(max2,in[a+4][b-4]);
       max2=max(max2,in[a+4][b-3]);
       max2=max(max2,in[a+4][b-2]);
       max2=max(max2,in[a+4][b-1]);
       max2=max(max2,in[a+4][b+0]);
       max2=max(max2,in[a+4][b+1]);
       max2=max(max2,in[a+4][b+2]);
       max2=max(max2,in[a+4][b+3]);
       max2=max(max2,in[a+4][b+4]);
       max2=max(max2,in[a+4][b+5]);
       max2=max(max2,in[a+4][b+6]);
       max2=max(max2,in[a+5][b-5]);
       max2=max(max2,in[a+5][b-4]);
       max2=max(max2,in[a+5][b-3]);
       max2=max(max2,in[a+5][b-2]);
       max2=max(max2,in[a+5][b-1]);
       max2=max(max2,in[a+5][b+0]);
       max2=max(max2,in[a+5][b+1]);
       max2=max(max2,in[a+5][b+2]);
       max2=max(max2,in[a+5][b+3]);
       max2=max(max2,in[a+5][b+4]);
       max2=max(max2,in[a+5][b+5]);
       max2=max(max2,in[a+6][b-4]);
       max2=max(max2,in[a+6][b-3]);
       max2=max(max2,in[a+6][b-2]);
       max2=max(max2,in[a+6][b-1]);
       max2=max(max2,in[a+6][b+0]);
       max2=max(max2,in[a+6][b+1]);
       max2=max(max2,in[a+6][b+2]);
       max2=max(max2,in[a+6][b+3]);
       max2=max(max2,in[a+6][b+4]);
       max2=max(max2,in[a+7][b-3]);
       max2=max(max2,in[a+7][b-2]);
       max2=max(max2,in[a+7][b-1]);
       max2=max(max2,in[a+7][b+0]);
       max2=max(max2,in[a+7][b+1]);
       max2=max(max2,in[a+7][b+2]);
       max2=max(max2,in[a+7][b+3]);    
       out[ig*H+jg]=max2; }
}

__global__ void kernel_copy_inv_char(BASE *in, BASE *out)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) out[i]=255-in[i];
}

__global__ void kernel_suma_float(BASE *in, BASE *in2, BASE *out)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) out[i]=(BASE)(in[i]+in2[i]);
}

// entrada=im, salida=tmp1
void tophat(BASE *im, BASE *tmp1, BASE *tmp2)
{  if(TOPHAT==0) 
   {  hipMemcpy(tmp1,im,H*V*sizeof(BASE),hipMemcpyDeviceToDevice); return;  }
   kernel_copy_inv_char<<<dimGrid1,dimBlock1>>>(im,tmp2);  
   if(TOPHAT==1)
   {  erosion_disk_st15_gpu<<<dimGrid,dimBlock>>>(tmp2,tmp1);
      dilate_disk_st15_gpu<<<dimGrid,dimBlock>>>(tmp1,tmp2); }
   else if(TOPHAT==2)
   {  erosion_st15_filas_win_gpu<<<dimGrid51,dimBlock>>>(tmp2,tmp1);
      erosion_st15_cols_win_gpu<<<dimGrid52,dimBlock>>>(tmp1,tmp2);
      dilate_st15_filas_win_gpu<<<dimGrid51,dimBlock>>>(tmp2,tmp1);
      dilate_st15_cols_win_gpu<<<dimGrid52,dimBlock>>>(tmp1,tmp2);  }
   else if(TOPHAT==3)
   {  erosion_st64_filas_win_gpu<<<dimGrid51a,dimBlocka>>>(tmp2,tmp1);
      erosion_st64_cols_win_gpu<<<dimGrid52a,dimBlockb>>>(tmp1,tmp2);
      dilate_st64_filas_win_gpu<<<dimGrid51a,dimBlocka>>>(tmp2,tmp1);
      dilate_st64_cols_win_gpu<<<dimGrid52a,dimBlockb>>>(tmp1,tmp2);  }
   kernel_suma_float<<<dimGrid1,dimBlock1>>>(im,tmp2,tmp1);  
}

// ------------------------------------------------------------------------
// PLS
// ------------------------------------------------------------------------
float s_cpd=+1; // s_cpd=-1 => disables the collision point detection
float k_ext=1;  // weight ext. potential (k_ext > 0 => evol. to 2QCNN black)
float k_int=1;  // weight the internal potential (k_int>0)
float k_inf=-1; // weight balloon potential: deflating (>0), inflating (<0)

__global__ void kernel1(float *int_pot, BASE *tmp1, float k)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) int_pot[i]=k*tmp1[i]; 
}

__global__ void kernel2(BASE *output, BASE *tmp1, BASE *output_GFE)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) output[i] = output[i]|(tmp1[i]&output_GFE[i]);
}

__global__ void kernel3(BASE *output, BASE *contour)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) output[i]=(1-output[i])|contour[i];
}

__global__ void kernel_inv_bin(BASE *in, float *out)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) out[i]=1-in[i];
}

__global__ void kernel_add_pot_inv(BASE *edge2, float max2, float *imd2, float kext, float *ext_pot)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) ext_pot[i]=kext*(100-(50*edge2[i]+50*(imd2[i]/max2)));
}

__global__ void kernel_distance_map(float *in, float *out, int k)
{ unsigned int i = threadIdx.x + BLOCK*BLOCK*blockIdx.x;
  if(i<H*V) out[i] += in[i]/k;
}

__global__ void f_CPDv3_gpu(const char dir1, BASE *output, BASE *mask_CPD, BASE boundary)
{  unsigned int i=threadIdx.x, j=threadIdx.y;
   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
   int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
   __shared__ BASE tmp1[BLOCK*BLOCK], tmp2[BLOCK*BLOCK], tmp3[BLOCK*BLOCK];

   if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) tmp2[i*BLOCK+j]=output[ig*H+jg];
   else tmp2[i*BLOCK+j]=boundary;
   __syncthreads();
  
   if(dir1=='N')
   {  if(i>0)tmp1[i*BLOCK+j]=tmp2[(i-1)*BLOCK+j]; else tmp1[i*BLOCK+j]=boundary;
      tmp3[i*BLOCK+j]=(1-tmp2[i*BLOCK+j])&tmp1[i*BLOCK+j];
      __syncthreads();
      if(j>0) tmp1[i*BLOCK+j]=tmp3[i*BLOCK+(j-1)]; else tmp1[i*BLOCK+j]=boundary;
      __syncthreads();
      if(j<BLOCK-1) tmp2[i*BLOCK+j]=tmp3[i*BLOCK+(j+1)]; else tmp2[i*BLOCK+j]=boundary; 
     __syncthreads(); }
   else if(dir1=='S')
   {  if(i<BLOCK-1) tmp1[i*BLOCK+j]=tmp2[(i+1)*BLOCK+j]; else tmp1[i*BLOCK+j]=boundary;
      tmp3[i*BLOCK+j]=(1-tmp2[i*BLOCK+j])&tmp1[i*BLOCK+j];
      __syncthreads();
      if(j>0) tmp1[i*BLOCK+j]=tmp3[i*BLOCK+(j-1)]; else tmp1[i*BLOCK+j]=boundary;
      __syncthreads();
      if(j<BLOCK-1) tmp2[i*BLOCK+j]=tmp3[i*BLOCK+(j+1)]; else tmp2[i*BLOCK+j]=boundary; 
      __syncthreads(); }
   else if(dir1=='E')
   {  if(j<BLOCK-1) tmp1[i*BLOCK+j]=tmp2[i*BLOCK+(j+1)]; else tmp1[i*BLOCK+j]=boundary;
      tmp3[i*BLOCK+j]=(1-tmp2[i*BLOCK+j])&tmp1[i*BLOCK+j];
      __syncthreads();
      if(i<BLOCK-1) tmp1[i*BLOCK+j]=tmp3[(i+1)*BLOCK+j]; else tmp1[i*BLOCK+j]=boundary;
      __syncthreads();
      if(i>0) tmp2[i*BLOCK+j]=tmp3[(i-1)*BLOCK+j]; else tmp2[i*BLOCK+j]=boundary;
      __syncthreads(); }
   else if(dir1=='W')
  {   if(j>0) tmp1[i*BLOCK+j]=tmp2[i*BLOCK+(j-1)]; else tmp1[i*BLOCK+j]=boundary;
      tmp3[i*BLOCK+j]=(1-tmp2[i*BLOCK+j])&tmp1[i*BLOCK+j];
      __syncthreads();
      if(i<BLOCK-1) tmp1[i*BLOCK+j]=tmp3[(i+1)*BLOCK+j]; else tmp1[i*BLOCK+j]=boundary;
      __syncthreads();
      if(i>0) tmp2[i*BLOCK+j]=tmp3[(i-1)*BLOCK+j]; else tmp2[i*BLOCK+j]=boundary;
      __syncthreads(); }

   if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H)) 
      mask_CPD[ig*H+jg]=tmp3[i*BLOCK+j]|tmp1[i*BLOCK+j]|tmp2[i*BLOCK+j];
}

__global__ void f_GFEv3_gpu(const char dir, float *ext_pot, float *int_pot, float kinf, BASE *inf_pot, BASE *mask_CPD, BASE *output_GFE, float boundary)
{  unsigned int i=threadIdx.x, j=threadIdx.y;
   int ig = threadIdx.x + (BLOCK-2)*blockIdx.x-1;
   int jg = threadIdx.y + (BLOCK-2)*blockIdx.y-1;
   __shared__ float aux1[BLOCK*BLOCK], aux2[BLOCK*BLOCK];
   
   if((ig>=0)&&(ig<V)&&(jg>=0)&&(jg<H)) 
        aux1[i*BLOCK+j]=ext_pot[ig*H+jg]+int_pot[ig*H+jg]+kinf*inf_pot[ig*H+jg]; 
   else aux1[i*BLOCK+j]=boundary;
   __syncthreads();
  
   if     ((dir=='S')&&(i>0))       aux2[i*BLOCK+j]=aux1[(i-1)*BLOCK+j];
   else if((dir=='W')&&(j<BLOCK-1)) aux2[i*BLOCK+j]=aux1[i*BLOCK+(j+1)];
   else if((dir=='N')&&(i<BLOCK-1)) aux2[i*BLOCK+j]=aux1[(i+1)*BLOCK+j];
   else if((dir=='E')&&(j>0))       aux2[i*BLOCK+j]=aux1[i*BLOCK+(j-1)];
   else aux2[i*BLOCK+j]=boundary; 

   if((i>0)&&(i<BLOCK-1)&&(j>0)&&(j<BLOCK-1)&&(ig<V)&&(jg<H)) 
   {  if((mask_CPD[ig*H+jg]==0)&&(aux1[i*BLOCK+j]-aux2[i*BLOCK+j]>=0.001)) output_GFE[ig*H+jg]=1; 
      else output_GFE[ig*H+jg]=0;  }
}

void pls(BASE *input, BASE *output, BASE *tmp1, BASE *mask_CPD, BASE *output_GFE, BASE *contour, float *ext_pot, float *int_pot, float *aux1, float *aux2, int iter)
{  int k, contador; BASE dir[4]={'N','E','S','W'}; char sentido;

   for(contador=0;contador<iter;contador++) 
   {  hipMemcpy(input,output,H*V*sizeof(BASE),hipMemcpyDeviceToDevice);
      
      for(sentido=+1;sentido>=-1;sentido=sentido-2)
      { f_edge_shared_gpu<<<dimGrid4,dimBlock4>>>(output,tmp1); 
        kernel1<<<dimGrid1,dimBlock1>>>(int_pot,tmp1,k_int);

        for(k=0;k<4;k++)
        {  if(s_cpd==-1) hipMemset(mask_CPD,0,H*V*sizeof(BASE));
           else {
             f_CPDv3_gpu<<<dimGrid4,dimBlock4>>>(dir[k], output, mask_CPD, 0); }
	     f_GFEv3_gpu<<<dimGrid4,dimBlock4>>>(dir[k],ext_pot,int_pot,sentido*k_inf,output,
	                                       mask_CPD,output_GFE,0);
           f_shift_global_gpu<<<dimGrid,dimBlock>>>(dir[k],output,tmp1,0);
	   kernel2<<<dimGrid1,dimBlock1>>>(output,tmp1,output_GFE);
        }
        f_edge_shared_gpu<<<dimGrid4,dimBlock4>>>(output,contour);
        kernel3<<<dimGrid1,dimBlock1>>>(output,contour);
      }
   }
}

int main()
{  pid_t f; int nimg; char fichero[60];
   collect_images(); 

   // LOAD IMAGES TO DISK CACHE OF LINUX
   f=fork();
   if(f==0) { for(nimg=0;nimg<NIMG;nimg++)
   {  sprintf(fichero,PREFIX,name[nimg]);
      read_image_ppm_cache(fichero); }
   } 
   else {  
   // COMPUTE SEGMENTATIONS
   BASE *img_cpu, *tmp_cpu;
   int k, nimg; float max2; struct timeval start, end, ini, fin;

   img_cpu=(BASE *)malloc(H*V*sizeof(BASE));
   tmp_cpu=(BASE *)malloc(H*V*sizeof(BASE));
   if((img_cpu==NULL)||(img_cpu==NULL)) { printf("no memoria\n"); exit(-1); }
  
   cuda_init_2_0();
   float *ext_pot, *int_pot, *aux1, *aux2;  // inf_pot es kinf*output 
   BASE *im, *input, *output, *mask_CPD, *output_GFE, *contour, *tmp1, *tmp2;
   hipMalloc((void**)&im,H*V*sizeof(BASE));
   hipMalloc((void**)&input,H*V*sizeof(BASE));
   hipMalloc((void**)&output,H*V*sizeof(BASE));
   hipMalloc((void**)&mask_CPD,H*V*sizeof(BASE));
   hipMalloc((void**)&output_GFE,H*V*sizeof(BASE));
   hipMalloc((void**)&contour,H*V*sizeof(BASE));
   hipMalloc((void**)&tmp1,H*V*sizeof(BASE));
   hipMalloc((void**)&tmp2,H*V*sizeof(BASE));
   hipMalloc((void**)&ext_pot,H*V*sizeof(float));
   hipMalloc((void**)&int_pot,H*V*sizeof(float));
   hipMalloc((void**)&aux1,H*V*sizeof(float));
   hipMalloc((void**)&aux2,H*V*sizeof(float));
   // reaprovechamos las variables para ahorrar memoria
   BASE *imt, *edge2; float *imd2;
   imt=mask_CPD; edge2=contour; imd2=int_pot;
 
   for(nimg=0;nimg<NIMG;nimg++)
   { // load data
     gettimeofday(&ini,NULL);
     printf("--------------------------------------\n");
     printf("* %s\n",name[nimg]); 
     sprintf(fichero,PREFIX,name[nimg]);
     read_image_ppm(fichero,img_cpu);
     hipMemcpy(im,img_cpu,H*V*sizeof(BASE),hipMemcpyHostToDevice);
     hipDeviceSynchronize(); gettimeofday(&start,NULL);

     // 1. Vessel pre-estimation
     tophat(im,tmp1,tmp2);
     diffusion_and_binarization(tmp1,imt,aux1,aux2,THRESHOLD);
     remove_outside_FOV_gpu<<<dimGrid1,dimBlock1>>>(imt,im,FOV_UMBRAL);
     remove_noise(imt,(int*)ext_pot,(int*)aux1,(int*)aux2,CONN,NOISE_SIZE);

     // 2. Initial contour estimation
     erosion_shared_gpu<<<dimGrid4,dimBlock4>>>(imt,output);
 
     // 3. External potential estimation
     // 3.1 Edge detection + logical operations
     sobel_edge_detect_shared_gpu<<<dimGrid4,dimBlock4>>>(imt,edge2);
     kernel_inv_bin<<<dimGrid1,dimBlock1>>>(edge2,imd2);
     // 3.2 Distance map
     for(k=1;k<=5;k++)
     {  erosion_float_shared_gpu<<<dimGrid4,dimBlock4>>>(imd2,aux2);
        kernel_distance_map<<<dimGrid1,dimBlock1>>>(aux2,imd2,k); }
     hipMemcpy(aux1,imd2,H*V*sizeof(float),hipMemcpyDeviceToDevice);
     filter_float_shared_gpu<<<dimGrid4,dimBlock4>>>(aux1,imd2,1);
     // max2=6.0;
     max3a<<<dimGrid1,dimBlock1>>>(imd2,aux1);
     max3b<<<dimGrid2,dimBlock2>>>(aux1);
     max3c<<<dimGrid3,dimBlock3>>>(aux1);
     hipMemcpy(&max2,aux1,sizeof(float),hipMemcpyDeviceToHost);
     kernel_add_pot_inv<<<dimGrid1,dimBlock1>>>(edge2,max2,imd2,k_ext,ext_pot);
 
     // 4. Contour evolution (PLS)
     pls(input,output,tmp1,mask_CPD,output_GFE,contour,ext_pot,int_pot,aux1,aux2,STAGES);
     
     // 5. Removes and postprocessings
     remove_outside_FOV_gpu<<<dimGrid1,dimBlock1>>>(output,im,FOV_UMBRAL); 
     remove_small_structures(output,(int*)ext_pot,(int*)aux1,(int*)aux2,CONN,SMALL_STRUC_SIZE);
     if(H>2000) postproc_big_images(output,tmp1);
     hipDeviceSynchronize(); gettimeofday(&end,NULL);
     hipMemcpy(img_cpu,output,H*V*sizeof(BASE),hipMemcpyDeviceToHost);

     // Save data
     hipMemcpy(im,img_cpu,H*V*sizeof(BASE),hipMemcpyHostToDevice);
     hipDeviceSynchronize(); 
     sprintf(fichero,"output/vessel_seg_gpu-%s.pgm",name[nimg]);
     save_image_pgm(fichero,img_cpu);
     hipDeviceSynchronize(); gettimeofday(&fin,NULL);
     t_tot[nimg]=(fin.tv_sec-ini.tv_sec)+(fin.tv_usec-ini.tv_usec)*1E-6;
     t_comp[nimg]=(end.tv_sec-start.tv_sec)+(end.tv_usec-start.tv_usec)*1E-6;
     compute_accuracies(img_cpu,tmp_cpu,nimg);
     printf(" t_comp=%f, t_total=%f\n",t_comp[nimg],t_tot[nimg]);
   }
   compute_averages();
   hipFree(im); hipFree(ext_pot); hipFree(input); hipFree(output);
   hipFree(int_pot); hipFree(mask_CPD); hipFree(output_GFE);
   hipFree(contour); hipFree(tmp1); hipFree(tmp2); hipFree(aux1); hipFree(aux2);
   free(img_cpu); free(tmp_cpu);
   cuda_error("Error in execution in GPU!!!"); 
   printf("Execution without exceptions in GPU\n");
   }
}

